#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>

#define POBLACION 100
#define LONG_COD 20
#define LIMITE -5.12
#define CROSS_PROBABILITY 0.3
#define MUTATION_PROBABILITY 0.001
#define INTERVALO 10.24/__powf(2,LONG_COD/2)

//#define BLOCKSIZE 128
#define BLOCKSIZE 8
#define WARPSIZE 32

typedef struct {
    char genotipo[LONG_COD];
    double aptitud;
} Individuo;

__device__ void decoder(double * x, double * y, char * genotipo) {
    int i;
    *x = *y = 0.0;

    // calculo del primer decimal
    for(i=0; i<LONG_COD/2; i++)
        *x += (int)(genotipo[i]) * __powf(2, (LONG_COD/2)-(i+1));
    *x = (*x) * INTERVALO + LIMITE;

    //calculo del segundo decimal
    for(;i<LONG_COD;i++)
        *y += (int)(genotipo[i]) * __powf(2, LONG_COD-(i+1));
    *y = (*y) * INTERVALO + LIMITE;
}

__host__ __device__ double fitness (double p1, double p2){
    return (p1*p1) + (p2*p2);
}

__global__
void tournamentSelectionKernel(Individuo * dev_poblacion, Individuo * dev_selection, hiprandState *dev_state){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx<POBLACION){
        hiprandState lstate = dev_state[idx];

        Individuo candidato_a, candidato_b;

        candidato_a = dev_poblacion[(int) (hiprand_uniform(&lstate)*(POBLACION-0.00001))];
        candidato_b = dev_poblacion[(int) (hiprand_uniform(&lstate)*(POBLACION-0.00001))];

        if (candidato_a.aptitud < candidato_b.aptitud)
            dev_selection[idx] = candidato_a;
        else
            dev_selection[idx] = candidato_b;

        dev_state[idx] = lstate;
    }
}

__device__
void sonMutationBlade(Individuo *sons, hiprandState *dev_state, int idx){
    int i,j;
    double randProbability;
    hiprandState lstate = dev_state[idx];
    for(i=0;i<2;i++)
        for(j=0;j<LONG_COD;j++)
            randProbability = (((double) LONG_COD)*hiprand_uniform(&lstate)*(POBLACION-0.00001));
            if(randProbability<MUTATION_PROBABILITY){
                if(sons[i].genotipo[j])
                    sons[i].genotipo[j]=0;
                else
                    sons[i].genotipo[j]=1;
            }
    dev_state[idx] = lstate;
}

__global__
void crossSelectionKernel(Individuo * dev_poblacion, Individuo * dev_selection, hiprandState *dev_state){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx<POBLACION-1){
        if(idx==0 || !idx%2){
            hiprandState lstate = dev_state[idx];
            double crossProbability = (((double) LONG_COD)*hiprand_uniform(&lstate)*(POBLACION-0.00001));
            if(crossProbability < MUTATION_PROBABILITY){
                int point, j, aux;
                double x, y;
                point = (int) (((double) LONG_COD)*hiprand_uniform(&lstate)*(POBLACION-0.00001));
                for(j=point; j<LONG_COD; j++){
                    aux=dev_selection[idx].genotipo[j];
                    dev_selection[idx].genotipo[j]=dev_selection[idx+1].genotipo[j];
                    dev_selection[idx+1].genotipo[j]=aux;
                }

                sonMutationBlade(&dev_selection[idx], dev_state, idx);

                decoder(&x, &y, dev_selection[idx].genotipo);
                dev_selection[idx].aptitud=fitness(x,y);

                decoder(&x, &y, dev_selection[idx+1].genotipo);
                dev_selection[idx+1].aptitud=fitness(x,y);
            }
            dev_state[idx] = lstate;
        }
    }
}

__inline__ __device__
double warpAllReduceCompare(double val) {
    for (int mask = WARPSIZE/2; mask > 0; mask /= 2){
        val = fmax(val,__shfl_down(val, mask));
    }
    return val;
}

__inline__ __device__
double blockReduceCompare(double val) {

    static __shared__ double shared[WARPSIZE]; // Shared mem for 32 partial sums
    int lane = threadIdx.x % WARPSIZE;
    int wid = threadIdx.x / WARPSIZE;

    val = warpAllReduceCompare(val);     // Each warp performs partial reduction

    if (lane==0) shared[wid]=val; // Write reduced value to shared memory
    __syncthreads();              // Wait for all partial reductions

    //read from shared memory only if that warp existed
    val = (threadIdx.x < blockDim.x / WARPSIZE) ? shared[lane] : 0;

    if (wid==0) val = warpReduceCompare(val); //Final reduce within first warp

    return val;
}

__global__
void eliteKernel(Individuo * dev_poblacion, Individuo * dev_selection, hiprandState *dev_state){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    double MAX = -1;
    for(;i<POBLACION;i+=blockDim.x * gridDim.x){
        MAX = MAX < dev_poblacion[i].aptitud ? dev_poblacion[i].aptitud : MAX;
    }

    MAX = blockReduceCompare(MAX);
    if(threadIdx.x==0)
        atomicMax();
}

__global__
void init_rand(hiprandState *dev_state, int seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < POBLACION)
        hiprand_init(idx + seed, 0, 0, &dev_state[idx]);
}

__global__
void init_poblacion(Individuo * dev_poblacion, hiprandState *dev_state){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < POBLACION){
        int i;
        double x, y;
        hiprandState lstate = dev_state[idx];
        for(i=0; i<LONG_COD; i++)
            dev_poblacion[idx].genotipo[i] = hiprand_uniform(&lstate) > 0.5 ? 1.0 : 0.0;
        decoder(&x, &y, dev_poblacion[idx].genotipo);
        dev_poblacion[idx].aptitud = fitness(x,y);
        dev_state[idx] = lstate;
    }
}

void print_selection(Individuo *host_seleccion);

int main (void) {
    srand(time(NULL));
    printf("[HOST] Starting script\n");

    int GRIDSIZE = (POBLACION+BLOCKSIZE-1)/BLOCKSIZE;
    dim3 block(BLOCKSIZE, 1, 1);
    dim3 grid(GRIDSIZE, 1, 1);

    int GRIDSIZE2 = (POBLACION/4+BLOCKSIZE-1)/BLOCKSIZE;
    dim3 grid2(GRIDSIZE2, 1, 1);

    printf("grid1 = %d; grid2 = %d\n", GRIDSIZE, GRIDSIZE2);

    /*
    * Random initialization.
    **/
    hiprandState *dev_state;
    hipMalloc(&dev_state, sizeof(hiprandState)*POBLACION);
    init_rand<<<grid, block>>>(dev_state, rand());

    Individuo * host_seleccion, * host_poblacion;
    Individuo * dev_seleccion, * dev_poblacion;

    host_poblacion = (Individuo *) malloc (sizeof(Individuo)*POBLACION);
    host_seleccion = (Individuo *) malloc (sizeof(Individuo)*POBLACION);
    hipMalloc((void**)&dev_poblacion, sizeof(Individuo)*POBLACION);
    hipMalloc((void**)&dev_seleccion, sizeof(Individuo)*POBLACION);

    init_poblacion<<<grid, block>>>(dev_poblacion, dev_state);
    tournamentSelectionKernel<<<grid, block>>>(dev_poblacion, dev_seleccion, dev_state);
    crossSelectionKernel<<<grid, block>>>(dev_poblacion, dev_seleccion, dev_state);
    eliteKernel<<<grid2, block>>>(dev_poblacion, dev_seleccion, dev_state);

    hipMemcpy(host_poblacion, dev_poblacion, sizeof(Individuo)*POBLACION, hipMemcpyDeviceToHost);
    hipMemcpy(host_seleccion, dev_seleccion, sizeof(Individuo)*POBLACION, hipMemcpyDeviceToHost);
    printf("\n");

    hipDeviceSynchronize();

    hipFree(dev_poblacion);
    hipFree(dev_seleccion);
    hipFree(dev_state);

    hipDeviceReset();
    return 0;
}

void print_selection(Individuo *host_seleccion){
    int i, j;
    for(i=0; i<POBLACION; i++){
        printf("\nhost_seleccion[%d] = %f", i, host_seleccion[i].aptitud);
        for(j=0; j<LONG_COD; j++){
            printf("\nhost_genotipo[%d] = %d", i, host_seleccion[i].genotipo[j]);
        }
    }

}
