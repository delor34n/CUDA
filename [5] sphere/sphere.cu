#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>

#define POBLACION 20
#define LONG_COD 20
#define LIMITE -5.12
#define PROB_CRUCE 0.3
#define PROB_MUTACION 0.001
#define INTERVALO 10.24/pow(2,LONG_COD/2)

#define nThreads 2
#define nBlocks 5

typedef struct {
    int genotipo[LONG_COD];
    double aptitud;
} Individuo;

/* PROC fitness (double p1, double p2) DEV (double)
 * MODIFICA nada
 * EFECTO recibe dos valores que representan los puntos que caracterizan a un individuo
 *  la funcion sirve para calcular la aptitud o fitness de cierto individuo segun sus
 *  puntos. este valor de aptitud es el que devuelve la funcion.
 */
__host__ __device__ double fitness (double p1, double p2){
    return pow(p1,2) + pow(p2,2);
}

__global__
void tournamentSelectionKernel(Individuo * dev_poblacion, Individuo * dev_selection, hiprandState *dev_state){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int rand_a = (int)truncf((((POBLACION/nThreads)+1)+0.999999)*hiprand_uniform(&dev_state[i]));
    int rand_b = (int)truncf((((POBLACION/nThreads)+1)+0.999999)*hiprand_uniform(&dev_state[i]));

    printf ("\nSELECTION[%d][%d]: dev_selection[%d] = %f", blockIdx.x, threadIdx.x, i, dev_poblacion[i].aptitud);
    if(i<POBLACION){
        Individuo candidato_a, candidato_b;

        candidato_a = dev_poblacion[rand_a];
        candidato_b = dev_poblacion[rand_b];

        if (candidato_a.aptitud < candidato_b.aptitud)
            dev_selection[i] = candidato_a;
        else
            dev_selection[i] = candidato_b;
        //printf ("\nSELECTION[%d][%d]: dev_selection[%d] = %f", blockIdx.x, threadIdx.x, i, dev_selection[i].aptitud);
    }
}

__global__
void crossSelectionKernel(Individuo * dev_selection){
    //int i = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("\nCROSS: dev_selection[%d]: %f", i, dev_selection[i].aptitud);
}

__global__
void eliteKernel(Individuo * dev_poblacion){
    //int i = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("\nELITE: dev_poblacion[%d]\n", i);
}

__global__
void init_rand(hiprandState *dev_state) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(1234, idx, 0, &dev_state[idx]);
}

Individuo generarIndividuo (void);
Individuo * generatePopulation (void);
Individuo init();

Individuo * fillSelection(void);
Individuo initFellow(void);

int generarBinario (void);
void decoder (double *, double *, int *);
void print_selection(Individuo *host_seleccion);

int main (void) {
    srand(time(NULL));
    printf("[HOST] Starting script\n");

    Individuo * host_seleccion = fillSelection(), * host_poblacion = generatePopulation();
    Individuo * dev_seleccion, * dev_poblacion;

    print_selection(host_poblacion);

    /*
    * Random initialization.
    **/
    hiprandState *dev_state;
    hipMalloc(&dev_state, nThreads*nBlocks);
    init_rand<<<nBlocks, nThreads>>>(dev_state);

    hipMalloc((void**)&dev_poblacion, sizeof(Individuo)*POBLACION);
    hipMalloc((void**)&dev_seleccion, sizeof(Individuo)*POBLACION/2);
    hipMemcpy(dev_poblacion, host_poblacion, sizeof(Individuo)*POBLACION, hipMemcpyHostToDevice);
    hipMemcpy(dev_seleccion, host_seleccion, sizeof(Individuo)*POBLACION/2, hipMemcpyHostToDevice);

    tournamentSelectionKernel<<<nBlocks, nThreads>>>(dev_poblacion, dev_seleccion, dev_state);
    hipMemcpy(host_seleccion, dev_seleccion, sizeof(Individuo)*POBLACION/2, hipMemcpyDeviceToHost);
    print_selection(host_seleccion);
    printf("\n");
    //crossSelectionKernel<<<nBlocks, nThreads>>>(dev_seleccion);

    hipDeviceSynchronize();

    hipFree(dev_poblacion);
    hipFree(dev_seleccion);
    hipFree(dev_state);

    hipDeviceReset();
    return 0;
}

void print_selection(Individuo *host_seleccion){
    int i;
    for(i=0; i<POBLACION; i++)
        printf("\nhost_seleccion[%d] = %f", i, host_seleccion[i].aptitud);
}

Individuo * fillSelection(void) {
    Individuo * aux;
    int i;

    aux = (Individuo *) malloc(sizeof(Individuo)*POBLACION/2);
    for(i=0;i<POBLACION/2;i++)
        aux[i] = initFellow();
    return aux;
}

Individuo initFellow(void){
    Individuo ind;
    int i;
    double x, y;

    for (i=0; i<LONG_COD; i++)
        ind.genotipo[i]=generarBinario();

    decoder(&x, &y, ind.genotipo);
    ind.aptitud = 0;

    return ind;
}

/* PROC generarPoblacion (void) DEV (Individuo *)
 * MODIFICA nada
 * EFECTO esta funcion genera una poblacion con la cantidad de individuos dada por la
 *  macro POBLACION. para generar cada individuo utiliza la funcion generarIndividuo()
 *  y una vez ha terminado el bucle, devuelve el puntero al primer individuo
 */
Individuo * generatePopulation(void) {
    Individuo * poblacion;
    int i;

    poblacion = (Individuo *) malloc(sizeof(Individuo)*POBLACION);
    for(i=0;i<POBLACION;i++)
        poblacion[i] = generarIndividuo();

    return poblacion;
}

/* PROC generarIndividuo (void) DEV (Individuo)
 * MODIFICA nada
 * EFECTO se encarga de generar un individuo utilizando valores aleatorios. primero crea
 *  la cadena de bits del genotipo utilizando la funcion generaBinario y luego evalua
 *  la aptitud del individuo utilizando la funcion decoder para decodificar el genotipo
 *  y la funcion fitness para obtener la aptitud.
 */
Individuo generarIndividuo (void){
    Individuo ind;
    int i;
    double x, y;

    for (i=0; i<LONG_COD; i++)
        ind.genotipo[i]=generarBinario();

    decoder(&x, &y, ind.genotipo);
    ind.aptitud = fitness(x,y);

    return ind;
}

/* PROC generarBinario (void) DEV (void)
 * MODIFICA nada
 * EFECTO se encarga de devolver un valor entero que siempre sera cero o uno. lo vamos a
 *  utilizar para generar los individuos al principio dado que su genoma es una cadena
 *  binaria que se genera aleatoriamente
 */
int generarBinario (void) {
    if (1 + (int) (10.0*rand()/(RAND_MAX+1.0)) > 5)
        return 1;
    else
        return 0;
}

/* PROC decoder (double *x, double *y, int *genotipo) DEV (double)
 * MODIFICA (double *x double *y)
 * EFECTO recibe un vector de enteros compuesto de 0 y 1 que representa dos numeros
 *  codificados en binario. se encarga de convertir estos dos numeros binarios a su
 *  equivalente en decimal con ayuda de la macro INTERVALO (incremento del valor entre
 *  cada binario) y la macro LIMITE que es el valor del limite inferior de la repre-
 *  sentacion que en el problema es -5.12
 */
void decoder (double * x, double * y, int * genotipo) {
    int i;
    *x = *y = 0.0;

    // calculo del primer decimal
    for(i=0; i<LONG_COD/2; i++)
        *x += genotipo[i] * pow(2, (LONG_COD/2)-(i+1));
    *x = (*x) * INTERVALO + LIMITE;

    //calculo del segundo decimal
    for(;i<LONG_COD;i++)
        *y += genotipo[i] * pow(2, LONG_COD-(i+1));
    *y = (*y) * INTERVALO + LIMITE;
}