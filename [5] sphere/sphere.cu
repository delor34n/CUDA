#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>

#define POBLACION 1000
#define LONG_COD 20
#define LIMITE -5.12
#define CROSS_PROBABILITY 0.3
#define MUTATION_PROBABILITY 0.001
#define INTERVALO 10.24/__powf(2,LONG_COD/2)

#define BLOCKSIZE 128

typedef struct {
    char genotipo[LONG_COD];
    double aptitud;
} Individuo;

__device__ void decoder(double * x, double * y, char * genotipo) {
    int i;
    *x = *y = 0.0;

    // calculo del primer decimal
    for(i=0; i<LONG_COD/2; i++)
        *x += (int)(genotipo[i]) * __powf(2, (LONG_COD/2)-(i+1));
    *x = (*x) * INTERVALO + LIMITE;

    //calculo del segundo decimal
    for(;i<LONG_COD;i++)
        *y += (int)(genotipo[i]) * __powf(2, LONG_COD-(i+1));
    *y = (*y) * INTERVALO + LIMITE;
}

__host__ __device__ double fitness (double p1, double p2){
    return (p1*p1) + (p2*p2);
}

__global__
void tournamentSelectionKernel(Individuo * dev_poblacion, Individuo * dev_selection, hiprandState *dev_state){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx<POBLACION){
        hiprandState lstate = dev_state[idx];

        Individuo candidato_a, candidato_b;

        candidato_a = dev_poblacion[(int) (hiprand_uniform(&lstate)*(POBLACION-0.00001))];
        candidato_b = dev_poblacion[(int) (hiprand_uniform(&lstate)*(POBLACION-0.00001))];

        if (candidato_a.aptitud < candidato_b.aptitud)
            dev_selection[idx] = candidato_a;
        else
            dev_selection[idx] = candidato_b;

        dev_state[idx] = lstate;
    }
}

__device__
void sonMutationBlade(Individuo *sons, hiprandState *dev_state, int idx){
    int i,j;
    double randProbability;
    hiprandState lstate = dev_state[idx];
    for(i=0;i<2;i++)
        for(j=0;j<LONG_COD;j++)
            randProbability = (((double) LONG_COD)*hiprand_uniform(&lstate)*(POBLACION-0.00001));
            if(randProbability<MUTATION_PROBABILITY){
                if(sons[i].genotipo[j])
                    sons[i].genotipo[j]=0;
                else
                    sons[i].genotipo[j]=1;
            }
    dev_state[idx] = lstate;
}

__global__
void crossSelectionKernel(Individuo * dev_poblacion, Individuo * dev_selection, hiprandState *dev_state){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx<POBLACION-1){
        if(idx==0 || !idx%2){
            hiprandState lstate = dev_state[idx];
            double crossProbability = (((double) LONG_COD)*hiprand_uniform(&lstate)*(POBLACION-0.00001));
            if(crossProbability < MUTATION_PROBABILITY){
                int point, j, aux;
                point = (int) (((double) LONG_COD)*hiprand_uniform(&lstate)*(POBLACION-0.00001));
                for(j=point; j<LONG_COD; j++){
                    aux=dev_selection[idx].genotipo[j];
                    dev_selection[idx].genotipo[j]=dev_selection[idx+1].genotipo[j];
                    dev_selection[idx+1].genotipo[j]=aux;
                }

                sonMutationBlade(&dev_selection[idx], dev_state, idx);
            }
            dev_state[idx] = lstate;
        }
    }
}

__global__
void eliteKernel(Individuo * dev_poblacion){
    //int i = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("\nELITE: dev_poblacion[%d]\n", i);
}

__global__
void init_rand(hiprandState *dev_state, int seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < POBLACION)
        hiprand_init(idx + seed, 0, 0, &dev_state[idx]);
}

__global__
void init_poblacion(Individuo * dev_poblacion, hiprandState *dev_state){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < POBLACION){
        int i;
        double x, y;
        hiprandState lstate = dev_state[idx];
        for(i=0; i<LONG_COD; i++)
            dev_poblacion[idx].genotipo[i] = hiprand_uniform(&lstate) > 0.5 ? 1.0 : 0.0;
        decoder(&x, &y, dev_poblacion[idx].genotipo);
        dev_poblacion[idx].aptitud = fitness(x,y);
        dev_state[idx] = lstate;
    }
}

void print_selection(Individuo *host_seleccion);

int main (void) {
    srand(time(NULL));
    printf("[HOST] Starting script\n");

    int GRIDSIZE = (POBLACION+BLOCKSIZE-1)/BLOCKSIZE;
    dim3 block(BLOCKSIZE, 1, 1);
    dim3 grid(GRIDSIZE, 1, 1);

    /*
    * Random initialization.
    **/
    hiprandState *dev_state;
    hipMalloc(&dev_state, sizeof(hiprandState)*POBLACION);
    init_rand<<<grid, block>>>(dev_state, rand());

    Individuo * host_seleccion, * host_poblacion;
    Individuo * dev_seleccion, * dev_poblacion;

    host_poblacion = (Individuo *) malloc (sizeof(Individuo)*POBLACION);
    host_seleccion = (Individuo *) malloc (sizeof(Individuo)*POBLACION);
    hipMalloc((void**)&dev_poblacion, sizeof(Individuo)*POBLACION);
    hipMalloc((void**)&dev_seleccion, sizeof(Individuo)*POBLACION);

    init_poblacion<<<grid, block>>>(dev_poblacion, dev_state);
    tournamentSelectionKernel<<<grid, block>>>(dev_poblacion, dev_seleccion, dev_state);
    crossSelectionKernel<<<grid, block>>>(dev_poblacion, dev_seleccion, dev_state);

    hipMemcpy(host_poblacion, dev_poblacion, sizeof(Individuo)*POBLACION, hipMemcpyDeviceToHost);
    hipMemcpy(host_seleccion, dev_seleccion, sizeof(Individuo)*POBLACION, hipMemcpyDeviceToHost);
    printf("\n");

    hipDeviceSynchronize();

    hipFree(dev_poblacion);
    hipFree(dev_seleccion);
    hipFree(dev_state);

    hipDeviceReset();
    return 0;
}

void print_selection(Individuo *host_seleccion){
    int i, j;
    for(i=0; i<POBLACION; i++){
        printf("\nhost_seleccion[%d] = %f", i, host_seleccion[i].aptitud);
        for(j=0; j<LONG_COD; j++){
            printf("\nhost_genotipo[%d] = %d", i, host_seleccion[i].genotipo[j]);
        }
    }

}
