#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>

#define POBLACION 100
#define LONG_COD 20
#define LIMITE -5.12
#define PROB_CRUCE 0.3
#define PROB_MUTACION 0.001
#define INTERVALO 10.24/pow(2,LONG_COD/2)

#define nThreads 10
#define nBlocks 10

typedef struct {
    int genotipo[LONG_COD];
    double aptitud;
} Individuo;

/* PROC fitness (double p1, double p2) DEV (double)
 * MODIFICA nada
 * EFECTO recibe dos valores que representan los puntos que caracterizan a un individuo
 *  la funcion sirve para calcular la aptitud o fitness de cierto individuo segun sus
 *  puntos. este valor de aptitud es el que devuelve la funcion.
 */
__host__ __device__ double fitness (double p1, double p2){
    return pow(p1,2) + pow(p2,2);
}

__global__
void tournamentSelectionKernel(Individuo * dev_poblacion, Individuo * dev_seleccion, hiprandState *dev_state){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int rand_a = (int)truncf((((POBLACION/nThreads)-1)+0.999999)*hiprand_uniform(&dev_state[i]));
    int rand_b = (int)truncf((((POBLACION/nThreads)-1)+0.999999)*hiprand_uniform(&dev_state[i]));
    printf("\nSELECTION[%d]: dev_poblacion[%d]: %f, RANDOM 1 = %d, RANDOM 2 = %d", threadIdx.x, i, dev_poblacion[i].aptitud, rand_a, rand_b);

    /*if(i<POBLACION){
        Individuo candidato_a, candidato_b;

        candidato_a = dev_poblacion[(int) (((double) POBLACION)*rand()/(RAND_MAX+1.0))];
        candidato_b = dev_poblacion[(int) (((double) POBLACION)*rand()/(RAND_MAX+1.0))];

        if (candidato_a.aptitud < candidato_b.aptitud)
            dev_seleccion[i] = candidato_a;
        else
            dev_seleccion[i] = candidato_b;
    }*/
}

__global__
void crossSelectionKernel(Individuo * dev_seleccion){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    printf("\nCROSS: antes dev_seleccion[%d]: %f", i, dev_seleccion[i].aptitud);
    dev_seleccion[i].aptitud = 3.14151617;
    printf("\nCROSS: despues dev_seleccion[%d]: %f", i, dev_seleccion[i].aptitud);
}

__global__
void eliteKernel(Individuo * dev_poblacion){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    printf("\nELITE: dev_poblacion[%d]\n", i);
}

__global__
void init_rand(hiprandState *dev_state) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(1234, idx, 0, &dev_state[idx]);
}

Individuo generarIndividuo (void);
Individuo * generarPoblacion (void);
Individuo init();

int generarBinario (void);
void decoder (double *, double *, int *);

int main (void) {
    srand(time(NULL));
    printf("[HOST] Starting script\n");

    Individuo * host_seleccion= generarPoblacion(), * host_poblacion = generarPoblacion();
    Individuo * dev_seleccion, * dev_poblacion;

    /*
    * Random initialization.
    **/
    hiprandState *dev_state;
    hipMalloc(&dev_state, nThreads*nBlocks);
    init_rand<<<nThreads,nBlocks>>>(dev_state);

    hipMalloc((void**)&dev_poblacion, sizeof(Individuo)*POBLACION);
    hipMalloc((void**)&dev_seleccion, sizeof(Individuo)*POBLACION);
    hipMemcpy(dev_poblacion, host_poblacion, sizeof(Individuo)*POBLACION, hipMemcpyHostToDevice);
    hipMemcpy(dev_seleccion, host_seleccion, sizeof(Individuo)*POBLACION, hipMemcpyHostToDevice);

    tournamentSelectionKernel<<<nThreads,nBlocks>>>(dev_poblacion, dev_seleccion, dev_state);
    hipMemcpy(dev_seleccion, host_seleccion, sizeof(Individuo)*POBLACION, hipMemcpyDeviceToHost);
    crossSelectionKernel<<<nThreads,nBlocks>>>(dev_seleccion);

    hipDeviceSynchronize();

    hipFree(dev_poblacion);
    hipFree(dev_seleccion);
    hipFree(dev_state);

    hipDeviceReset();
    return 0;
}

/* PROC generarPoblacion (void) DEV (Individuo *)
 * MODIFICA nada
 * EFECTO esta funcion genera una poblacion con la cantidad de individuos dada por la
 *  macro POBLACION. para generar cada individuo utiliza la funcion generarIndividuo()
 *  y una vez ha terminado el bucle, devuelve el puntero al primer individuo
 */
Individuo * generarPoblacion(void) {
    Individuo * poblacion;
    int i;

    poblacion = (Individuo *) malloc(sizeof(Individuo)*POBLACION);
    for(i=0;i<POBLACION;i++)
        poblacion[i] = generarIndividuo();

    return poblacion;
}

/* PROC generarIndividuo (void) DEV (Individuo)
 * MODIFICA nada
 * EFECTO se encarga de generar un individuo utilizando valores aleatorios. primero crea
 *  la cadena de bits del genotipo utilizando la funcion generaBinario y luego evalua
 *  la aptitud del individuo utilizando la funcion decoder para decodificar el genotipo
 *  y la funcion fitness para obtener la aptitud.
 */
Individuo generarIndividuo (void){
    Individuo ind;
    int i;
    double x, y;

    for (i=0; i<LONG_COD; i++)
        ind.genotipo[i]=generarBinario();

    decoder(&x, &y, ind.genotipo);
    ind.aptitud = fitness(x,y);

    return ind;
}

/* PROC generarBinario (void) DEV (void)
 * MODIFICA nada
 * EFECTO se encarga de devolver un valor entero que siempre sera cero o uno. lo vamos a
 *  utilizar para generar los individuos al principio dado que su genoma es una cadena
 *  binaria que se genera aleatoriamente
 */
int generarBinario (void) {
    if (1 + (int) (10.0*rand()/(RAND_MAX+1.0)) > 5)
        return 1;
    else
        return 0;
}

/* PROC decoder (double *x, double *y, int *genotipo) DEV (double)
 * MODIFICA (double *x double *y)
 * EFECTO recibe un vector de enteros compuesto de 0 y 1 que representa dos numeros
 *  codificados en binario. se encarga de convertir estos dos numeros binarios a su
 *  equivalente en decimal con ayuda de la macro INTERVALO (incremento del valor entre
 *  cada binario) y la macro LIMITE que es el valor del limite inferior de la repre-
 *  sentacion que en el problema es -5.12
 */
void decoder (double * x, double * y, int * genotipo) {
    int i;
    *x = *y = 0.0;

    // calculo del primer decimal
    for(i=0; i<LONG_COD/2; i++)
        *x += genotipo[i] * pow(2, (LONG_COD/2)-(i+1));
    *x = (*x) * INTERVALO + LIMITE;

    //calculo del segundo decimal
    for(;i<LONG_COD;i++)
        *y += genotipo[i] * pow(2, LONG_COD-(i+1));
    *y = (*y) * INTERVALO + LIMITE;
}