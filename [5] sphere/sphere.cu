#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>

#define POBLACION 600000
#define LONG_COD 20
#define LIMITE -5.12
#define PROB_CRUCE 0.3
#define PROB_MUTACION 0.001
#define INTERVALO 10.24/__powf(2,LONG_COD/2)

#define BLOCKSIZE 128

typedef struct {
    char genotipo[LONG_COD];
    double aptitud;
} Individuo;

__device__ void decoder(double * x, double * y, char * genotipo) {
    int i;
    *x = *y = 0.0;

    // calculo del primer decimal
    for(i=0; i<LONG_COD/2; i++)
        *x += (int)(genotipo[i]) * __powf(2, (LONG_COD/2)-(i+1));
    *x = (*x) * INTERVALO + LIMITE;

    //calculo del segundo decimal
    for(;i<LONG_COD;i++)
        *y += (int)(genotipo[i]) * __powf(2, LONG_COD-(i+1));
    *y = (*y) * INTERVALO + LIMITE;
}

__host__ __device__ double fitness (double p1, double p2){
    return (p1*p1) + (p2*p2);
}

__global__
void tournamentSelectionKernel(Individuo * dev_poblacion, Individuo * dev_selection, hiprandState *dev_state){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;    
    if(idx<POBLACION){
        hiprandState lstate = dev_state[idx];
        
        Individuo candidato_a, candidato_b;
        int aux = (int) (hiprand_uniform(&lstate)*(POBLACION-0.00001));
        printf("\n %d", aux);
        candidato_a = dev_poblacion[aux];
        
        aux = (int) (hiprand_uniform(&lstate)*(POBLACION-0.00001));
        printf("\n %d", aux);
        candidato_b = dev_poblacion[aux];

        if (candidato_a.aptitud < candidato_b.aptitud)
            dev_selection[idx] = candidato_a;
        else
            dev_selection[idx] = candidato_b;
    }
}

__global__
void crossSelectionKernel(Individuo * dev_selection){
    //int i = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("\nCROSS: dev_selection[%d]: %f", i, dev_selection[i].aptitud);
}

__global__
void eliteKernel(Individuo * dev_poblacion){
    //int i = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("\nELITE: dev_poblacion[%d]\n", i);
}

__global__
void init_rand(hiprandState *dev_state, int seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < POBLACION)
        hiprand_init(idx + seed, 0, 0, &dev_state[idx]);
}

__global__
void init_poblacion(Individuo * dev_poblacion, hiprandState *dev_state){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < POBLACION){
        int i;
        double x, y;
        hiprandState lstate = dev_state[idx];
        for(i=0; i<LONG_COD; i++)
            dev_poblacion[idx].genotipo[i] = hiprand_uniform(&lstate) > 0.5 ? 1.0 : 0.0;
        decoder(&x, &y, dev_poblacion[idx].genotipo);
        dev_poblacion[idx].aptitud = fitness(x,y);
        dev_state[idx] = lstate;
    }
}

void print_selection(Individuo *host_seleccion);

int main (void) {
    srand(time(NULL));
    printf("[HOST] Starting script\n");

    int GRIDSIZE = (POBLACION+BLOCKSIZE-1)/BLOCKSIZE;
    dim3 block(BLOCKSIZE, 1, 1);
    dim3 grid(GRIDSIZE, 1, 1);

    /*
    * Random initialization.
    **/
    hiprandState *dev_state;
    hipMalloc(&dev_state, sizeof(hiprandState)*POBLACION);
    init_rand<<<grid, block>>>(dev_state, rand());

    Individuo * host_seleccion, * host_poblacion;
    Individuo * dev_seleccion, * dev_poblacion;

    host_poblacion = (Individuo *) malloc (sizeof(Individuo)*POBLACION);
    host_seleccion = (Individuo *) malloc (sizeof(Individuo)*POBLACION);
    hipMalloc((void**)&dev_poblacion, sizeof(Individuo)*POBLACION);
    hipMalloc((void**)&dev_seleccion, sizeof(Individuo)*POBLACION);

    init_poblacion<<<grid, block>>>(dev_poblacion, dev_state);
    tournamentSelectionKernel<<<grid, block>>>(dev_poblacion, dev_seleccion, dev_state);

    hipMemcpy(host_poblacion, dev_poblacion, sizeof(Individuo)*POBLACION, hipMemcpyDeviceToHost);
    hipMemcpy(host_seleccion, dev_seleccion, sizeof(Individuo)*POBLACION, hipMemcpyDeviceToHost);
    //print_selection(host_poblacion);
    //print_selection(host_seleccion);
    printf("\n");
    
    //tournamentSelectionKernel<<<nBlocks, nThreads>>>(dev_poblacion, dev_seleccion, dev_state);
    //crossSelectionKernel<<<nBlocks, nThreads>>>(dev_seleccion);

    hipDeviceSynchronize();

    hipFree(dev_poblacion);
    hipFree(dev_seleccion);
    hipFree(dev_state);

    hipDeviceReset();
    return 0;
}

void print_selection(Individuo *host_seleccion){
    int i, j;
    for(i=0; i<POBLACION; i++){
        printf("\nhost_seleccion[%d] = %f", i, host_seleccion[i].aptitud);
        for(j=0; j<LONG_COD; j++){
            printf("\nhost_genotipo[%d] = %d", i, host_seleccion[i].genotipo[j]);
        }
    }

}