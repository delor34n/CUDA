
#include <hip/hip_runtime.h>
#include <stdio.h>

#define SIZE 12

#define nThreads 2
#define nBlocks 6

__global__ void splitBladeKernel(int * dev_a){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    printf("\n[DEVICE; BLOCK:%d; THREAD:%d] dev_a[%d] = %d",blockIdx.x, threadIdx.x, idx, dev_a[idx]);
    __syncthreads();
}

void fill_array(int *host_a);
void show_array(int *host_a);

int main (void){
    printf("[HOST] InitBlade\n");

    int * host_a = (int *) malloc(sizeof(int)*SIZE);
    fill_array(host_a);
    show_array(host_a);
    printf("\n");

    int * dev_a;
    hipMalloc((void**)&dev_a, sizeof(int)*SIZE);
    hipMemcpy(dev_a, host_a, sizeof(int)*SIZE, hipMemcpyHostToDevice);

    splitBladeKernel<<<nBlocks,nThreads>>>(dev_a);

    hipDeviceSynchronize();
    hipFree(dev_a);

    printf("\n");
    return 0;
}

void fill_array(int *host_a){
    int i;
    for(i=0; i<SIZE; i++)
        host_a[i] = i + 1;
}

void show_array(int *host_a){
    int i;
    for(i=0; i<SIZE; i++)
        printf("\nhost_a[%d] = %d",i, host_a[i]);
}