
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

int main(void) {
    //Cantidad de elementos que tendra el array.
    int num_elements = 16;
    //Se calcula la cantidad de bytes que se utilizara en total (int=4bytes => 16x4=64bytes)
    int num_bytes = num_elements * sizeof(int);

    //Un puntero para la GPU
    int *device_array = 0;
    //Un puntero para la CPU
    int *host_array = 0;

    //Se realiza el allocate del host (CPU)
    host_array = (int*)malloc(num_bytes);

    //Se realiza el allocate del device (GPU)
    hipMalloc((void**)&device_array, num_bytes);

    //Llenamos de cero el array que se encuentra en la GPU
    hipMemset(device_array, 12, num_bytes);

    //Copiamos el contenido del array que se encuentra en la GPU en el array de la CPU
    hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);

    //Imprimimos cada elemento del array del host (CPU)
    for(int i = 0; i < num_elements; ++i)
        printf("%d ", host_array[i]);

    //Liberamos la memoria solicitada para el host
    free(host_array);

    //Liberamos la memoria solicitada para el device
    hipFree(device_array);

    return 0;
}
