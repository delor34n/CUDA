#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 26
#define BLOCKSIZE 8
#define WARPSIZE 32
#define uint64 unsigned long long

typedef struct {
    char id[1];
    double num;
} Dato;

__inline__ __device__
double warpAllReduceCompare(double val) {
    for (unsigned int mask = WARPSIZE/2; mask > 0; mask /= 2){
        val = fmax(val,__shfl_down(__double2float_rd(val), mask, WARPSIZE));
    }
    return val;
}

__inline__ __device__
double blockReduceCompare(double val) {
    static __shared__ double shared[WARPSIZE]; // Shared mem for 32 partial sums
    int lane = threadIdx.x % WARPSIZE;
    int wid = threadIdx.x / WARPSIZE;

    val = warpAllReduceCompare(val);     // Each warp performs partial reduction

    if (lane==0) shared[wid]=val; // Write reduced value to shared memory
        __syncthreads();              // Wait for all partial reductions

    //read from shared memory only if that warp existed
    val = (threadIdx.x < blockDim.x / WARPSIZE) ? shared[lane] : 0;

    if (wid==0) val = warpAllReduceCompare(val); //Final reduce within first warp
        return val;
}


__device__ void AtomicMax(double * const address, const double value) {
    if (* address >= value)
        return;
    uint64 * const address_as_i = (uint64 *)address;
    uint64 old = * address_as_i, assumed;
    do {
        assumed = old;
        if (__longlong_as_double(assumed) >= value)
            break;
        old = atomicCAS(address_as_i, assumed, __double_as_longlong(value));
    } while (assumed != old);
}

__global__
void eliteKernel(Dato * device_datos){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int k = i;
    double MAX = -1;
    for(;i<N;i+=blockDim.x * gridDim.x){
        MAX = MAX < device_datos[i].num ? device_datos[i].num : MAX;
    }

    MAX = blockReduceCompare(MAX);
    if(threadIdx.x & (warpSize-1)==0)
        AtomicMax(&device_datos[k].num,MAX);
}

void fill(Dato *host_datos);

int main(void){
    srand(time(NULL));
    printf("[HOST] STARTING SCRIPT\n");

    Dato *host_datos;
    Dato *device_datos;

    host_datos = (Dato *) malloc (sizeof(Dato)*N);
    device_datos = (Dato *) malloc (sizeof(Dato)*N);
    hipMalloc((void**)&device_datos, sizeof(Dato)*N);

    fill(host_datos);

    int GRIDSIZE = (N/4+BLOCKSIZE-1)/BLOCKSIZE;
    dim3 block(BLOCKSIZE, 1, 1);
    dim3 grid(GRIDSIZE, 1, 1);

    hipMemcpy(host_datos, device_datos, sizeof(Dato)*N, hipMemcpyHostToDevice);
    eliteKernel<<<grid,block>>>(device_datos);
    hipMemcpy(device_datos, host_datos, sizeof(Dato)*N, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipDeviceReset();

    printf("\nMAX = %s[%f]\n\n", host_datos[0].id, host_datos[0].num);
    printf("[HOST] SCRIPT EXECUTION FINISHED\n");
    return 0;
}

void fill(Dato *host_datos){
    int i;
    int letter=97;
    for(i=0;i<N;i++,letter++){
        host_datos[i].id[0]=(char)letter;
        host_datos[i].num=(double)(rand()/(RAND_MAX/(99.0-0.1)));

        printf("%s:%f\n", host_datos[i].id, host_datos[i].num);
    }
}
