#include "hip/hip_runtime.h"
#include <stdio.h>
#include <curl/curl.h>

#define ATTACKS 1000
#define BLOCKSIZE 128

__host__ __device__ void curl_run(){
    CURL *curl;
    curl = curl_easy_init();
}

__global__
void curlKernel(){
    curl_run();
}

int main(){
    printf("[HOST] STARTING SCRIPT\n");

    int GRIDSIZE = (ATTACKS+BLOCKSIZE-1)/BLOCKSIZE;
    dim3 block(BLOCKSIZE, 1, 1);
    dim3 grid(GRIDSIZE, 1, 1);

    curlKernel<<<grid,block>>>();
    hipDeviceSynchronize();
    hipDeviceReset();

    printf("[HOST] SCRIPT EXECUTION FINISHED\n");
    return 0;
}
